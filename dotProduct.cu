
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <sys/time.h>

const int threads_per_block = 512;

// Forward function declarations
float GPU_big_dot(float *A, float *B, int N, float *time);
float GPU_big_dot2(float *A, float *B, int N, float *time);
float *get_random_vector(int N);
void die(const char *message);

int main(int argc, char **argv) {
	// Seed the random generator (use a constant here for repeatable results)
	srand(10);

	// Determine the vector length
	int N = 1000;  // default value
	//int N = 1 << 24;  // default value
	if (argc > 1) N = atoi(argv[1]); // user-specified value

	// Generate two random vectors
	float *A = get_random_vector(N);
	float *B = get_random_vector(N);
	
	// Compute their dot product using GPU1 kernal
	float GPU1_Time[1] = {0};
	float GPU1 = GPU_big_dot(A, B, N, GPU1_Time);
	
	// Compute their dot product using GPU2 kernal
	float GPU2_Time[1] = {0};
	float GPU2 = GPU_big_dot2(A, B, N, GPU2_Time);
	
	// Compute the speedup or slowdown
	if (GPU1_Time[0] > GPU2_Time[0]) printf("\nThe kernal with atomics outperformed the kernal without it by %.2fx\n", GPU1_Time[0] / GPU2_Time[0]);
	else                     printf("\nThe kernal without atomics outperformed the kernal with it by %.2fx\n", GPU2_Time[0] / GPU1_Time[0]);
	
	// Check the correctness of the GPU results
        if (fabs(GPU1 - GPU2) > 0.000001) 
	  printf("\nvalues incorrect, CPU dot product = %f, GPU dot product = %f\n", GPU1, GPU2);
	else           
	  printf("\nvalues correct, CPU dot product = %f, GPU dot product = %f\n", GPU1, GPU2);

}

// A GPU kernel that computes the vector dot product of A and B
// (uses shared mem and parallel reduction)
__global__ void dot_product_kernel1(float *a, float *b, float *out, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
    int block_diff = blockDim.x*gridDim.x;

    __shared__ float cache[threads_per_block];

    //float temp = 0.0f; /////this is the issue why its off/////////////////////
    while(index < n) {
        cache[threadIdx.x] += a[index] * b[index] ;
        index += block_diff;
    }

    //cache[threadIdx.x] = temp;

    __syncthreads();

    unsigned int i = blockDim.x/2;
    while(i != 0) {
        if(threadIdx.x < i) {
            cache[threadIdx.x] += cache[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

	if(threadIdx.x == 0) out[blockIdx.x] = cache[0];
}

// Returns the vector dot product of A and B
// Calls kernal1
float GPU_big_dot(float *A_CPU, float *B_CPU, int N, float *time) {	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// Allocate GPU memory for the inputs and the result
	int vector_size = N * sizeof(float);
	float *A_GPU, *B_GPU, *GPU2;
	if (hipMalloc((void **) &A_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	if (hipMalloc((void **) &B_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	if (hipMalloc((void **) &GPU2, vector_size) != hipSuccess) die("Error allocating GPU memory");
	
	// Transfer the input vectors to GPU memory
	hipMemcpy(A_GPU, A_CPU, vector_size, hipMemcpyHostToDevice);
	hipMemcpy(B_GPU, B_CPU, vector_size, hipMemcpyHostToDevice);
		
	// Determine the number of thread blocks in the grid 
	int blocks_per_grid = (int) ((float) (N + threads_per_block - 1) / (float) threads_per_block);
	
	// Execute the kernel to compute the vector dot product on the GPU
	hipEventRecord(start);
	dot_product_kernel1<<< blocks_per_grid , threads_per_block >>> (A_GPU, B_GPU, GPU2, N);
	hipDeviceSynchronize(); 
	hipEventRecord(stop);
	
	
	// Check for kernel errors
	hipError_t error = hipGetLastError();
	if (error) {
	  char message[256];
	  sprintf(message, "CUDA error: %s", hipGetErrorString(error));
	  die(message);
	}
	
	// Allocate CPU memory for the result
	float *GPU1 = (float *) malloc(vector_size);
	if (GPU1 == NULL) die("Error allocating CPU memory");
	
	// Transfer the result from the GPU to the CPU
	hipMemcpy(GPU1, GPU2, vector_size, hipMemcpyDeviceToHost);

	// Get time
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	time[0] = milliseconds;
	
	// Free the GPU memory
	hipFree(A_GPU);
	hipFree(B_GPU);
	hipFree(GPU2);
	
	//Do summation of multiplication in CPU
	float sum = 0;
	for(int i = 0; i < N; i++) sum += GPU1[i];

	return sum;
}

// A GPU kernel that computes the vector dot product of A and B
// (uses shared mem, parallel reduction, and atomics)
__global__ void dot_product_kernel2(float *a, float *b, float *out, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
    int block_diff = blockDim.x*gridDim.x;

    __shared__ float cache[threads_per_block];

    float temp = 0.0f; /////this is the issue why its off/////////////////////
    while(index < n) {
        temp += a[index] * b[index] ;
        index += block_diff;
    }

    cache[threadIdx.x] = temp;

    __syncthreads();

    unsigned int i = blockDim.x/2;
    while(i != 0) {
        if(threadIdx.x < i) {
            cache[threadIdx.x] += cache[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }
    
    if(threadIdx.x == 0) {
        atomicAdd(out, cache[0]);
    }
}

// Returns the vector dot product of A and B (computed on the GPU)
// Calls kernal2
float GPU_big_dot2(float *A_CPU, float *B_CPU, int N, float time[]) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// Allocate GPU memory for the inputs and the result
	int vector_size = N * sizeof(float);
	float *A_GPU, *B_GPU, *GPU2;
	if (hipMalloc((void **) &A_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	if (hipMalloc((void **) &B_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	if (hipMalloc((void **) &GPU2, vector_size) != hipSuccess) die("Error allocating GPU memory");
	
	// Transfer the input vectors to GPU memory
	hipMemcpy(A_GPU, A_CPU, vector_size, hipMemcpyHostToDevice);
	hipMemcpy(B_GPU, B_CPU, vector_size, hipMemcpyHostToDevice);
		
	// Determine the number of thread blocks in the grid 
	int blocks_per_grid = (int) ((float) (N + threads_per_block - 1) / (float) threads_per_block);
	
	// Execute the kernel to compute the vector dot product on the GPU
	hipEventRecord(start);
	dot_product_kernel2<<< blocks_per_grid , threads_per_block >>> (A_GPU, B_GPU, GPU2, N);
	hipDeviceSynchronize(); 
	hipEventRecord(stop);
	
	// Check for kernel errors
	hipError_t error = hipGetLastError();
	if (error) {
	  char message[256];
	  sprintf(message, "CUDA error: %s", hipGetErrorString(error));
	  die(message);
	}
	
	// Allocate CPU memory for the result
	float *GPU1 = (float *) malloc(vector_size);
	if (GPU1 == NULL) die("Error allocating CPU memory");
	
	// Transfer the result from the GPU to the CPU
	hipMemcpy(GPU1, GPU2, vector_size, hipMemcpyDeviceToHost);

	// Get time
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	time[0] = milliseconds;
	
	// Free the GPU memory
	hipFree(A_GPU);
	hipFree(B_GPU);
	hipFree(GPU2);
	
	//return sum from kernal
	return GPU1[0];
}

// Returns a randomized vector containing N elements
float *get_random_vector(int N) {
	if (N < 1) die("Number of elements must be greater than zero");
	
	// Allocate memory for the vector
	float *V = (float *) malloc(N * sizeof(float));
	if (V == NULL) die("Error allocating CPU memory");
	
	// Populate the vector with random numbers
	for (int i = 0; i < N; i++) V[i] = (float) rand() / (float) rand();
	
	// Return the randomized vector
	return V;
}

// Prints the specified message and quits
void die(const char *message) {
	printf("%s\n", message);
	exit(1);
}